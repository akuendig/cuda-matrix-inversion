#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../include/helper.h"

#define N 30

int main(int argc, char *argv[]) {
    float a[N];
    float *dev_a;
    int pivot, i;
    hipblasHandle_t handle;

    /* Pre-processing steps */
    gpuErrchk( hipMalloc((void **) &dev_a, N * sizeof(float)) );

    /* Input column major matrix */
    for(i = 0; i < N; i++)
        scanf("%f", &a[N]);
    gpuErrchk( hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice) );

    /* Invert the matrix */
    cublasErrchk( hipblasCreate(&handle) );
    cublasErrchk( hipblasIsamax(handle,
                    N,          // Number of elements to be searched
                    dev_a,      // Starting position
                    1,          // Increment in words (NOT BYTES)
                    &pivot)     // Maximum element in the col
    );

    printf("%d\n", pivot);

    /* Cleanup the mess */
    cublasErrchk( hipblasDestroy(handle) );
    gpuErrchk( hipFree(dev_a) );

    return 0;
}
