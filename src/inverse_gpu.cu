#include "hip/hip_runtime.h"
#include <stdio.h>	
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define SWAP(x, y, z)	((z) = (x),(x) = (y),(y) = (z))

#define N 5
#define	DataType	float
#define ArraySize	(N * N * sizeof(DataType))

void printMatrix(float a[N][N]) {
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++)
			printf("%f\t", a[i][j]);
		printf("\n");
	}
	printf("\n");
}

void pivotRow(hipblasHandle_t &handle, DataType *a, DataType *a_inv, int col) {
	int pivot = -1;

	hipblasIsamax(handle,
		N - col,					// Number of elements to be searched
		a + (col * N) + col,		// Starting position
		1,							// Increment in words (NOT BYTES)
		&pivot);					// Maximum element in the col
	int row = pivot - (col * N);	// Row number with maximum element

	hipblasSswap(handle,
		N,							// Nuber of elements to be swapped
		a + col,					// Current row
		N,							// Increment (becuase of column major)
		a + row,					// Row with max pivot
		N);
	hipblasSswap(handle, N, a_inv + col, N, a_inv + row, N);
}

void normalizeRow(hipblasHandle_t &handle, DataType *a, DataType *a_inv, int row) {
	DataType scalar = a[row * N + row];

	hipblasSscal(handle, N, &scalar, a + row, N);
	hipblasSscal(handle, N, &scalar, a_inv + row, N);
}

__global__
void transform_matrix(DataType *a DataType *a_inv, int row) {
	__shared__ DataType scalars[N];

	// store the scalars corresponding to the column 'row'
	scalars[threadIdx.x] = a[row * N + threadIdx.x];
	__syncthreads();

	// No need to transform 'row'th row
	if(threadIdx.x == row)
		return;

	// Each thread transforms column
	DataType pivot_elem = a[threadIdx.x * N + row];
	for(int i = 0; i < N; i++) {
		a[threadIdx.x * N + i] -= (scalars[threadIdx.x] * pivot_elem);
	}
}

void invert(hipblasHandle_t &handle, DataType *a, DataType *a_inv) {
	for(int i = 0; i < N; i++) {
		// Pivot the matrix
		pivotRow(handle, a, a_inv, i);

		// Make column entry to be one
		normalizeRow(a, a_inv, i);

		// Number of threads equals number of rows
		transform_matrix<<<1, N>>>(a, a_inv, i);
	}
}

int main(int argc, char *argv[]) {
	DataType *a, *a_inv;
	DataType *dev_a, **dev_a_inv;
	hipblasHandle_t handle;

	/* Pre-processing steps */
	if(!(a = (DataType *)malloc(ArraySize))) {
		perror("");
		return errno;
	}
	if(!(a_inv = (DataType *)malloc(ArraySize))) {
		perror("");
		return errno;
	}
	hipMalloc(&dev_a, ArraySize);
	hipMalloc(&dev_a_inv, ArraySize);
	hipblasCreate(&handle);

	/* Input column major matrix */
	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++) {
			scanf("%f", &a[i * N + j])
			if(i == j)
				a_inv[i * N + j] = 1;
			else
				a_inv[i * N + j] = 0;
		}
	hipMemcpy(dev_a, a, ArraySize, hipMemcpyHostToDevice);
	hipMemcpy(dev_a_inv, a_inv, ArraySize, hipMemcpyHostToDevice);

	/* Invert the matrix */
	invert(handle, dev_a, dev_a_inv);

	/* Display the result */
	hipMemcpy(a_inv, dev_a_inv, ArraySize, hipMemcpyDeviceToHost);
	printf("Inverse is:\n");
	printMatrix(a_inv);

	/* Cleanup the mess */
	free(a);
	free(a_inv);
	hipFree(dev_a);
	hipFree(dev_a_inv);
	hipblasDestroy(handle);

	return 0;
}
