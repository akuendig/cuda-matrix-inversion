#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <errno.h>

#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "../include/helper.h"

#define ELEMENT_TYPE float

void benchmarkMalloc(const int numReplications, const int numElems) {
    const size_t sizeOfData = sizeof(ELEMENT_TYPE)*numElems;
    ELEMENT_TYPE *data = (ELEMENT_TYPE *)malloc(sizeOfData);
    float timeOfMallocSum, timeOfFreeSum;

    printf("Benchmark MALLOC - Replications: %d Elements: %d\n", numReplications, numElems);

    hipEvent_t start, stop;
    gpuErrchk( hipEventCreate(&start) );
    gpuErrchk( hipEventCreate(&stop) );

    float *devData;

    for (int i = 0; i < numReplications; ++i) {
        gpuErrchk( hipEventRecord(start) );
        gpuErrchk( hipMalloc((void**)&devData, sizeOfData) );
        gpuErrchk( hipEventRecord(stop) );

        gpuErrchk( hipEventSynchronize(stop) );
        float timeOfMalloc = 0;
        gpuErrchk( hipEventElapsedTime(&timeOfMalloc, start, stop) );

        gpuErrchk( hipEventRecord(start) );
        gpuErrchk( hipFree(devData) );
        gpuErrchk( hipEventRecord(stop) );

        gpuErrchk( hipEventSynchronize(stop) );
        float timeOfFree = 0;
        gpuErrchk( hipEventElapsedTime(&timeOfFree, start, stop) );

        printf("Benchmark MALLOC - malloc: %fms free: %fms\n", timeOfMalloc, timeOfFree);

        timeOfMallocSum += timeOfMalloc;
        timeOfFreeSum += timeOfFree;
    }

    printf("Benchmark MALLOC - malloc: %f\n", (timeOfMallocSum/float(numReplications)));
    printf("Benchmark MALLOC - free: %f\n", (timeOfFreeSum/float(numReplications)));

    free(data);
}

void benchmarkTransfer(const int numReplications, const int numElems) {
    const size_t sizeOfData = sizeof(ELEMENT_TYPE)*numElems;
    ELEMENT_TYPE *data = (ELEMENT_TYPE *)malloc(sizeOfData);
    float timeToDeviceSum, timeFromDeviceSum;

    printf("Benchmark TRANSFER - Replications: %d Elements: %d\n", numReplications, numElems);

    hipEvent_t start, stop;
    gpuErrchk( hipEventCreate(&start) );
    gpuErrchk( hipEventCreate(&stop) );

    float *devData;

    gpuErrchk( hipMalloc((void**)&devData, sizeOfData) );

    for (int i = 0; i < numReplications; ++i) {
        gpuErrchk( hipEventRecord(start) );
        gpuErrchk( hipMemcpy(devData, data, sizeOfData, hipMemcpyHostToDevice) );
        gpuErrchk( hipEventRecord(stop) );

        gpuErrchk( hipEventSynchronize(stop) );
        float timeToDevice = 0;
        gpuErrchk( hipEventElapsedTime(&timeToDevice, start, stop) );

        gpuErrchk( hipEventRecord(start) );
        gpuErrchk( hipMemcpy(data, devData, sizeOfData, hipMemcpyDeviceToHost) );
        gpuErrchk( hipEventRecord(stop) );

        gpuErrchk( hipEventSynchronize(stop) );
        float timeFromDevice = 0;
        gpuErrchk( hipEventElapsedTime(&timeFromDevice, start, stop) );

        printf("Benchmark TRANSFER - To: %fms From: %fms\n", timeToDevice, timeFromDevice);

        timeToDeviceSum += timeToDevice;
        timeFromDeviceSum += timeFromDevice;
    }

    printf("Benchmark TRANSFER - Bandwidth to Device (GB/s): %f\n", sizeOfData/(timeToDeviceSum/float(numReplications))/1e6);
    printf("Benchmark TRANSFER - Bandwidth from Device (GB/s): %f\n", sizeOfData/(timeFromDeviceSum/float(numReplications))/1e6);

    gpuErrchk( hipFree(devData) );
    free(data);
};

void benchmarkTransferPitched(const int numReplications, const int numElems) {
    const size_t sizeOfData = sizeof(ELEMENT_TYPE)*numElems;
    ELEMENT_TYPE *data;
    float timeToDeviceSum, timeFromDeviceSum;

    printf("Benchmark TRANSFER PITCHED - Replications: %d Elements: %d\n", numReplications, numElems);

    hipEvent_t start, stop;
    gpuErrchk( hipEventCreate(&start) );
    gpuErrchk( hipEventCreate(&stop) );

    float *devData;

    gpuErrchk( hipHostAlloc((void**)&data, sizeOfData, hipHostMallocDefault) );
    gpuErrchk( hipMalloc((void**)&devData, sizeOfData) );

    for (int i = 0; i < numReplications; ++i) {
        gpuErrchk( hipEventRecord(start) );
        gpuErrchk( hipMemcpy(devData, data, sizeOfData, hipMemcpyHostToDevice) );
        gpuErrchk( hipEventRecord(stop) );

        gpuErrchk( hipEventSynchronize(stop) );
        float timeToDevice = 0;
        gpuErrchk( hipEventElapsedTime(&timeToDevice, start, stop) );

        gpuErrchk( hipEventRecord(start) );
        gpuErrchk( hipMemcpy(data, devData, sizeOfData, hipMemcpyDeviceToHost) );
        gpuErrchk( hipEventRecord(stop) );

        gpuErrchk( hipEventSynchronize(stop) );
        float timeFromDevice = 0;
        gpuErrchk( hipEventElapsedTime(&timeFromDevice, start, stop) );

        printf("Benchmark TRANSFER PITCHED - To: %fms From: %fms\n", timeToDevice, timeFromDevice);

        timeToDeviceSum += timeToDevice;
        timeFromDeviceSum += timeFromDevice;
    }

    printf("Benchmark TRANSFER PITCHED - Bandwidth to Device (GB/s): %f\n", sizeOfData/(timeToDeviceSum/float(numReplications))/1e6);
    printf("Benchmark TRANSFER PITCHED - Bandwidth from Device (GB/s): %f\n", sizeOfData/(timeFromDeviceSum/float(numReplications))/1e6);

    gpuErrchk( hipFree(devData) );
    free(data);
};

void benchmarkTransfer2D(const int numReplications, const int numElems, const int numArrays) {
    const size_t sizeOfData = sizeof(ELEMENT_TYPE)*numElems;
    ELEMENT_TYPE *data = (ELEMENT_TYPE *)malloc(sizeOfData);
    float timeToDeviceSum, timeFromDeviceSum;

    printf("Benchmark TRANSFER 2D - Replications: %d Elements: %d Arrays: %d\n", numReplications, numElems, numArrays);

    hipEvent_t start, stop;
    gpuErrchk( hipEventCreate(&start) );
    gpuErrchk( hipEventCreate(&stop) );

    float *devData;
    size_t pitch;

    gpuErrchk( hipHostAlloc((void**)&data, sizeOfData*numArrays, hipHostMallocDefault) );
    gpuErrchk( hipMallocPitch((void**)&devData, &pitch, sizeOfData, numArrays) );

    for (int i = 0; i < numReplications; ++i) {
        gpuErrchk( hipEventRecord(start) );
        gpuErrchk( hipMemcpy2D(devData, pitch, data, sizeOfData, sizeOfData, numArrays,
               hipMemcpyHostToDevice) );
        gpuErrchk( hipEventRecord(stop) );

        gpuErrchk( hipEventSynchronize(stop) );
        float timeToDevice = 0;
        gpuErrchk( hipEventElapsedTime(&timeToDevice, start, stop) );

        gpuErrchk( hipEventRecord(start) );
        gpuErrchk( hipMemcpy2D(data, sizeOfData, devData, pitch, sizeOfData, numArrays,
               hipMemcpyDeviceToHost) );
        gpuErrchk( hipEventRecord(stop) );

        gpuErrchk( hipEventSynchronize(stop) );
        float timeFromDevice = 0;
        gpuErrchk( hipEventElapsedTime(&timeFromDevice, start, stop) );

        printf("Benchmark TRANSFER 2D - To: %fms From: %fms\n", timeToDevice, timeFromDevice);

        timeToDeviceSum += timeToDevice;
        timeFromDeviceSum += timeFromDevice;
    }

    printf("Benchmark TRANSFER 2D - Bandwidth to Device (GB/s): %f\n", sizeOfData/(timeToDeviceSum/float(numReplications))/1e6);
    printf("Benchmark TRANSFER 2D - Bandwidth from Device (GB/s): %f\n", sizeOfData/(timeFromDeviceSum/float(numReplications))/1e6);

    gpuErrchk( hipFree(devData) );
    free(data);
};

int main(int argc, char const *argv[])
{
    ensure(argc >= 4, "Usage: %s NUM_REPLICATIONS NUM_ELEMENTS NUM_ARRAYS", argv[0]);

    int numReplications = atoi(argv[1]);
    int numElems = atoi(argv[2]);
    int numArrays = atoi(argv[3]);

    benchmarkMalloc(numReplications, numElems);
    benchmarkTransfer(numReplications, numElems);
    benchmarkTransferPitched(numReplications, numElems);
    benchmarkTransfer2D(numReplications, numElems, numArrays);

    /* code */
    return 0;
}
