#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <errno.h>

#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "../include/helper.h"

#define ELEMENT_TYPE float

// int benchmarkMalloc(const int numAllocs, const int numElems) {

// }

void benchmarkTransfer(const int numReplications, const int numElems) {
    const size_t sizeOfData = sizeof(ELEMENT_TYPE)*numElems;
    ELEMENT_TYPE *data = (ELEMENT_TYPE *)malloc(sizeOfData);
    float timeToDeviceSum, timeFromDeviceSum;

    printf("Benchmark TRANSFER - Replications: %d Elements: %d\n", numReplications, numElems);

    hipEvent_t start, stop;
    gpuErrchk( hipEventCreate(&start) );
    gpuErrchk( hipEventCreate(&stop) );

    float *devData;

    gpuErrchk( hipMalloc((void**)&devData, sizeOfData) );

    for (int i = 0; i < numReplications; ++i) {
        gpuErrchk( hipEventRecord(start) );
        gpuErrchk( hipMemcpy(devData, data, sizeOfData, hipMemcpyHostToDevice) );
        gpuErrchk( hipEventRecord(stop) );

        gpuErrchk( hipEventSynchronize(stop) );
        float timeToDevice = 0;
        gpuErrchk( hipEventElapsedTime(&timeToDevice, start, stop) );

        gpuErrchk( hipEventRecord(start) );
        gpuErrchk( hipMemcpy(data, devData, sizeOfData, hipMemcpyDeviceToHost) );
        gpuErrchk( hipEventRecord(stop) );

        gpuErrchk( hipEventSynchronize(stop) );
        float timeFromDevice = 0;
        gpuErrchk( hipEventElapsedTime(&timeFromDevice, start, stop) );

        printf("Benchmark TRANSFER - To: %f From: %f\n", timeToDevice, timeFromDevice);

        timeToDeviceSum += timeToDevice;
        timeFromDeviceSum += timeFromDevice;
    }

    printf("Benchmark TRANSFER - Bandwidth to Device (GB/s): %f\n", sizeOfData/(timeToDeviceSum/float(numReplications))/1e6);
    printf("Benchmark TRANSFER - Bandwidth from Device (GB/s): %f\n", sizeOfData/(timeFromDeviceSum/float(numReplications))/1e6);

    gpuErrchk( hipFree(devData) );
    free(data);
};

int main(int argc, char const *argv[])
{
    benchmarkTransfer(3, 1024*1024*200);
    /* code */
    return 0;
}
