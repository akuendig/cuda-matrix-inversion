#include "hip/hip_runtime.h"
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../../include/types.h"
#include "../../include/helper.h"
#include "../../include/inverse.h"

#define SWAP(x, y, z)   ((z) = (x),(x) = (y),(y) = (z))


void pivotRow(hipblasHandle_t &handle, Array a, Array a_inv, int col, int N) {
    int pivot = -1;

    cublasErrchk( hipblasIsamax(handle,
        N - col,            // Number of elements to be searched
        a + (col * N) + col,        // Starting position
        1,              // Increment in words (NOT BYTES)
        &pivot) );            // Maximum element in the col
    int row = pivot - 1 + col;          // Row number with maximum element (starts with 1)

    // printf("Pivot: %d\nRow: %d\n", pivot, row);
    if(row == col)
        return;

    cublasErrchk( hipblasSswap(handle,
        N,              // Nuber of elements to be swapped
        a + col,            // Current row
        N,              // Increment (becuase of column major)
        a + row,            // Row with max pivot
        N) );
    cublasErrchk( hipblasSswap(handle, N, a_inv + col, N, a_inv + row, N) );
}

void normalizeRow(hipblasHandle_t handle, Array a, Array a_inv, int row, int N) {
    DataType scalar;

    gpuErrchk( hipMemcpy(&scalar, &a[row * N + row], sizeof(DataType), hipMemcpyDeviceToHost) );
    scalar = 1 / scalar;
    cublasErrchk( hipblasSscal(handle, N, &scalar, a + row, N) );
    cublasErrchk( hipblasSscal(handle, N, &scalar, a_inv + row, N) );
}

__global__
void transform_matrix(Array a, Array a_inv, int row, int N) {
    __shared__ DataType scalars[64];
    __shared__ DataType currRowA[64], currRowI[64];

    // store the scalars corresponding to the column 'row'
    scalars[threadIdx.x] = a[row * N + threadIdx.x];
    currRowA[threadIdx.x] = a[threadIdx.x * N + row];
    currRowI[threadIdx.x] = a_inv[threadIdx.x * N + row];
    __syncthreads();

    // No need to transform 'row'th row
    if(threadIdx.x == row)
        return;

    // Each thread transforms row
    for(int i = 0; i < N; i++) {
        a[i * N + threadIdx.x] -= (scalars[threadIdx.x] * currRowA[i]);
        a_inv[i * N + threadIdx.x] -= (scalars[threadIdx.x] * currRowI[i]);
    }
}

void invert(hipblasHandle_t &handle, Array devA, Array devAInv, int N) {
    for(int i = 0; i < N; i++) {
        // Pivot the matrix
        pivotRow(handle, devA, devAInv, i, N);

        // Make column entry to be one
        normalizeRow(handle, devA, devAInv, i, N);

        // Number of threads equals number of rows
        transform_matrix<<<1, N>>>(devA, devAInv, i, N);

        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    }
}

__global__
void inverse_gauss_kernel(Array *a, Array *aInv, int N) {
    int row, pivot;
    hipblasHandle_t handle;

    hipblasCreate(&handle);

    for (row = 0; row < N; ++row) {
        /*cublasErrchk*/( hipblasIsamax(handle,
            N - row,            // Number of elements to be searched
            &a[threadIdx.x][(row * N) + row],        // Starting position
            1,              // Increment in words (NOT BYTES)
            &pivot) );            // Maximum element in the row
        int pivotRow = pivot - 1 + row;          // Row number with maximum element (starts with 1)

        // printf("Pivot: %d\nRow: %d\n", pivot, pivotRow);
        if(pivotRow != row) {
            /*cublasErrchk*/( hipblasSswap(handle,
                N,              // Nuber of elements to be swapped
                &a[threadIdx.x][row],            // Current pivotRow
                N,              // Increment (becuase of column major)
                &a[threadIdx.x][pivotRow],            // Row with max pivot
                N) );
            /*cublasErrchk*/( hipblasSswap(handle,
                N,
                &aInv[threadIdx.x][row],
                N,
                &aInv[threadIdx.x][pivotRow],
                N) );
        }

        DataType scalar = 1/a[threadIdx.x][row * N + row];

        /*cublasErrchk*/( hipblasSscal(handle,
            N,
            &scalar,
            &a[threadIdx.x][row],
            N) );
        /*cublasErrchk*/( hipblasSscal(handle,
            N,
            &scalar,
            &aInv[threadIdx.x][row],
            N) );

        transform_matrix<<<1, N>>>(a[threadIdx.x], aInv[threadIdx.x], row, N);
    }

    hipblasDestroy(handle);
}

// Allocates one continous array of memory of size arraySize*batchSize and writes the
// pointers of all subarrays into the array of pointers located at devArrayPtr.
static hipError_t batchedCudaMalloc(Array* devArrayPtr, size_t *pitch, size_t arraySize, int batchSize) {
    char *devPtr;

    hipError_t result = hipMallocPitch((void**)&devPtr, pitch, arraySize, batchSize);

    if (hipSuccess != result) {
        return result;
    }

    for (int i = 0; i < batchSize; ++i) {
        devArrayPtr[i] = (Array)devPtr;
        devPtr += *pitch;
    }

    return hipSuccess;
}

extern "C" void inverse_gauss_kernel_gpu(
        hipblasHandle_t handle,
        int n,
        Array As,
        Array aInvs,
        int batchSize) {

    int k, i;
    Array *devAs;
    size_t pitchAs;
    Array *devAInvs;
    size_t pitchAInvs;

    const size_t ArraySize = sizeof(DataType) * n * n;

    gpuErrchk( hipHostAlloc((void**)&devAs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devAInvs, sizeof(Array)*batchSize, hipHostMallocDefault) );

    gpuErrchk( batchedCudaMalloc(devAs, &pitchAs, ArraySize, batchSize) );
    gpuErrchk( batchedCudaMalloc(devAInvs, &pitchAInvs, ArraySize, batchSize) );

    memset(aInvs, 0, batchSize*ArraySize);

    for (k = 0; k < batchSize; ++k) {
        for (i = 0; i < n; ++i) {
            aInvs[k*n*n + i*n + i] = 1.f;
        }
    }

    gpuErrchk( hipMemcpy2D(devAs[0], pitchAs, As, ArraySize, ArraySize, batchSize,
                hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy2D(devAInvs[0], pitchAInvs, aInvs, ArraySize, ArraySize, batchSize,
                hipMemcpyHostToDevice) );

    inverse_gauss_kernel<<<1, batchSize>>>(devAs, devAInvs, n);

    gpuErrchk( hipMemcpy2D(aInvs, ArraySize, devAInvs[0], pitchAInvs, ArraySize, batchSize,
                hipMemcpyDeviceToHost) );
    gpuErrchk( hipFree((void*)devAs[0]) );
    gpuErrchk( hipFree((void*)devAInvs[0]) );
    gpuErrchk( hipHostFree((void*)devAs) );
    gpuErrchk( hipHostFree((void*)devAInvs) );
}

// int main(int argc, char *argv[]) {
//  Array a, a_inv;
//  Array dev_a, dev_a_inv;
//  hipblasHandle_t handle;

//  /* Pre-processing steps */
//  if(!(a = (Array)malloc(ArraySize))) {
//      perror("");
//      return errno;
//  }
//  if(!(a_inv = (Array)malloc(ArraySize))) {
//      perror("");
//      return errno;
//  }
//  hipMalloc(&dev_a, ArraySize);
//  hipMalloc(&dev_a_inv, ArraySize);
//  hipblasCreate(&handle);

//  /* Input column major matrix */
//  for(int i = 0; i < N; i++)
//      for(int j = 0; j < N; j++) {
//          scanf("%f", &a[i * N + j]);
//          if(i == j)
//              a_inv[i * N + j] = 1;
//          else
//              a_inv[i * N + j] = 0;
//      }
//  hipMemcpy(dev_a, a, ArraySize, hipMemcpyHostToDevice);
//  hipMemcpy(dev_a_inv, a_inv, ArraySize, hipMemcpyHostToDevice);

//  /* Invert the matrix */
//  invert(handle, dev_a, dev_a_inv);

//  /* Display the result */
//  hipMemcpy(a, dev_a, ArraySize, hipMemcpyDeviceToHost);
//  hipMemcpy(a_inv, dev_a_inv, ArraySize, hipMemcpyDeviceToHost);
//  printf("Inverse is:\n");
//  //printMatrix(a);
//  printMatrix(a_inv);

//  /* Cleanup the mess */
//  free(a);
//  free(a_inv);
//  hipFree(dev_a);
//  hipFree(dev_a_inv);
//  hipblasDestroy(handle);

//  return 0;
// }
