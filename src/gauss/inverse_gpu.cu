#include "hip/hip_runtime.h"
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../../include/types.h"
#include "../../include/helper.h"
#include "../../include/inverse.h"

#define SWAP(x, y, z)   ((z) = (x),(x) = (y),(y) = (z))


void pivotRow(hipblasHandle_t &handle, Array a, Array a_inv, int col, int N) {
    int pivot = -1;

    cublasErrchk( hipblasIsamax(handle,
        N - col,            // Number of elements to be searched
        a + (col * N) + col,        // Starting position
        1,              // Increment in words (NOT BYTES)
        &pivot) );            // Maximum element in the col
    int row = pivot - 1 + col;          // Row number with maximum element (starts with 1)

    // printf("Pivot: %d\nRow: %d\n", pivot, row);
    if(row == col)
        return;

    cublasErrchk( hipblasSswap(handle,
        N,              // Nuber of elements to be swapped
        a + col,            // Current row
        N,              // Increment (becuase of column major)
        a + row,            // Row with max pivot
        N) );
    cublasErrchk( hipblasSswap(handle, N, a_inv + col, N, a_inv + row, N) );
}

void normalizeRow(hipblasHandle_t handle, Array a, Array a_inv, int row, int N) {
    DataType scalar;

    gpuErrchk( hipMemcpy(&scalar, &a[row * N + row], sizeof(DataType), hipMemcpyDeviceToHost) );
    scalar = 1 / scalar;
    cublasErrchk( hipblasSscal(handle, N, &scalar, a + row, N) );
    cublasErrchk( hipblasSscal(handle, N, &scalar, a_inv + row, N) );
}

__global__
void transform_matrix(Array a, Array a_inv, int row, int N) {
    __shared__ DataType scalars[64];
    __shared__ DataType currRowA[64], currRowI[64];

    // store the scalars corresponding to the column 'row'
    scalars[threadIdx.x] = a[row * N + threadIdx.x];
    currRowA[threadIdx.x] = a[threadIdx.x * N + row];
    currRowI[threadIdx.x] = a_inv[threadIdx.x * N + row];
    __syncthreads();

    // No need to transform 'row'th row
    if(threadIdx.x == row)
        return;

    // Each thread transforms row
    for(int i = 0; i < N; i++) {
        a[i * N + threadIdx.x] -= (scalars[threadIdx.x] * currRowA[i]);
        a_inv[i * N + threadIdx.x] -= (scalars[threadIdx.x] * currRowI[i]);
    }
}

void invert(hipblasHandle_t &handle, Array devA, Array devAInv, int N) {
    for(int i = 0; i < N; i++) {
        // Pivot the matrix
        pivotRow(handle, devA, devAInv, i, N);

        // Make column entry to be one
        normalizeRow(handle, devA, devAInv, i, N);

        // Number of threads equals number of rows
        transform_matrix<<<1, N>>>(devA, devAInv, i, N);

        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    }
}

__global__
void inverse_gauss_kernel(Array a, Array aInv, int N) {
    int row, pivot;
    hipblasHandle_t handle;

    hipblasCreate(&handle);

    for (row = 0; row < N; ++row) {
        /*cublasErrchk*/( hipblasIsamax(handle,
            N - row,            // Number of elements to be searched
            a + (row * N) + row,        // Starting position
            1,              // Increment in words (NOT BYTES)
            &pivot) );            // Maximum element in the row
        int pivotRow = pivot - 1 + row;          // Row number with maximum element (starts with 1)

        // printf("Pivot: %d\nRow: %d\n", pivot, pivotRow);
        if(pivotRow != row) {
            /*cublasErrchk*/( hipblasSswap(handle,
                N,              // Nuber of elements to be swapped
                a + row,            // Current pivotRow
                N,              // Increment (becuase of column major)
                a + pivotRow,            // Row with max pivot
                N) );
            /*cublasErrchk*/( hipblasSswap(handle,
                N,
                aInv + row,
                N,
                aInv + pivotRow,
                N) );
        }

        DataType scalar = 1/a[row * N + row];

        /*cublasErrchk*/( hipblasSscal(handle,
            N,
            &scalar,
            a + row,
            N) );
        /*cublasErrchk*/( hipblasSscal(handle,
            N,
            &scalar,
            aInv + row,
            N) );

        transform_matrix<<<1, N>>>(a, aInv, row, N);
    }

    hipblasDestroy(handle);
}

extern "C" void inverse_gauss_gpu(Array a, int n) {
    int i;
    Array aInv, devA, devAInv;

    const size_t ArraySize = n*n * sizeof(DataType);
    aInv = (Array)malloc(ArraySize);
    ensure(aInv, "could not allocate 0x%lX bytes of memory for matrix inverse", ArraySize);

    memset(aInv, 0, ArraySize);
    for (i = 0; i < n; ++i) { aInv[i*n + i] = 1.f; }

    gpuErrchk( hipMalloc(&devA, ArraySize) );
    gpuErrchk( hipMalloc(&devAInv, ArraySize) );

    gpuErrchk( hipMemcpy(devA, a, ArraySize, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(devAInv, aInv, ArraySize, hipMemcpyHostToDevice) );

    /* Invert the matrix */
    inverse_gauss_kernel<<<1, 1>>>(devA, devAInv, n);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    /* Display the result */
    gpuErrchk( hipMemcpy(a, devAInv, ArraySize, hipMemcpyDeviceToHost) );

    /* Cleanup the mess */
    gpuErrchk( hipFree(devAInv) );
    gpuErrchk( hipFree(devA) );

    free(aInv);
}

// int main(int argc, char *argv[]) {
//  Array a, a_inv;
//  Array dev_a, dev_a_inv;
//  hipblasHandle_t handle;

//  /* Pre-processing steps */
//  if(!(a = (Array)malloc(ArraySize))) {
//      perror("");
//      return errno;
//  }
//  if(!(a_inv = (Array)malloc(ArraySize))) {
//      perror("");
//      return errno;
//  }
//  hipMalloc(&dev_a, ArraySize);
//  hipMalloc(&dev_a_inv, ArraySize);
//  hipblasCreate(&handle);

//  /* Input column major matrix */
//  for(int i = 0; i < N; i++)
//      for(int j = 0; j < N; j++) {
//          scanf("%f", &a[i * N + j]);
//          if(i == j)
//              a_inv[i * N + j] = 1;
//          else
//              a_inv[i * N + j] = 0;
//      }
//  hipMemcpy(dev_a, a, ArraySize, hipMemcpyHostToDevice);
//  hipMemcpy(dev_a_inv, a_inv, ArraySize, hipMemcpyHostToDevice);

//  /* Invert the matrix */
//  invert(handle, dev_a, dev_a_inv);

//  /* Display the result */
//  hipMemcpy(a, dev_a, ArraySize, hipMemcpyDeviceToHost);
//  hipMemcpy(a_inv, dev_a_inv, ArraySize, hipMemcpyDeviceToHost);
//  printf("Inverse is:\n");
//  //printMatrix(a);
//  printMatrix(a_inv);

//  /* Cleanup the mess */
//  free(a);
//  free(a_inv);
//  hipFree(dev_a);
//  hipFree(dev_a_inv);
//  hipblasDestroy(handle);

//  return 0;
// }
