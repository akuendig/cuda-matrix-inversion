#include "hip/hip_runtime.h"
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../../include/types.h"
#include "../../include/helper.h"

#define SWAP(x, y, z)   ((z) = (x),(x) = (y),(y) = (z))

#define N 64
#define ArraySize   (N * N * sizeof(DataType))

void pivotRow(hipblasHandle_t handle, DataType *a, DataType *a_inv, int col) {
	int pivot = -1;

	hipblasIsamax(handle,
		N - col,				// Number of elements to be searched
		a + (col * N) + col,	// Starting position
		1,						// Increment in words (NOT BYTES)
		&pivot);				// Maximum element in the col
	int row = pivot - 1 + col;	// Row number with maximum element (starts with 1)

	// printf("Pivot: %d\nRow: %d\n", pivot, row);
	if(row == col)
		return;

	hipblasSswap(handle,
		N,					// Nuber of elements to be swapped
		a + col,			// Current row
		N,					// Increment (becuase of column major)
		a + row,			// Row with max pivot
		N);
	hipblasSswap(handle, N, a_inv + col, N, a_inv + row, N);
}

void normalizeRow(hipblasHandle_t handle, Array a, Array a_inv, int row) {
    DataType scalar;

    gpuErrchk( hipMemcpy(&scalar, &a[row * N + row], sizeof(DataType), hipMemcpyDeviceToHost) );
    scalar = 1 / scalar;
    cublasErrchk( hipblasSscal(handle, N, &scalar, a + row, N) );
    cublasErrchk( hipblasSscal(handle, N, &scalar, a_inv + row, N) );
}

__global__
void transform_matrix(Array a, Array a_inv, int row) {
    __shared__ DataType scalars[N];
    __shared__ DataType currRowA[N], currRowI[N];

    // store the scalars corresponding to the column 'row'
    scalars[threadIdx.x] = a[row * N + threadIdx.x];
    currRowA[threadIdx.x] = a[threadIdx.x * N + row];
    currRowI[threadIdx.x] = a_inv[threadIdx.x * N + row];
    __syncthreads();

    // No need to transform 'row'th row
    if(threadIdx.x == row)
        return;

    // Each thread transforms row
    for(int i = 0; i < N; i++) {
        a[i * N + threadIdx.x] -= (scalars[threadIdx.x] * currRowA[i]);
        a_inv[i * N + threadIdx.x] -= (scalars[threadIdx.x] * currRowI[i]);
    }
}

void invert(hipblasHandle_t handle, Array devA, Array devAInv) {
    for(int i = 0; i < N; i++) {
        // Pivot the matrix
        pivotRow(handle, devA, devAInv, i);

        // Make column entry to be one
        normalizeRow(handle, devA, devAInv, i);

        // Number of threads equals number of rows
        transform_matrix<<<1, N>>>(devA, devAInv, i);

        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    }
}

extern "C" void inverse_gauss_gpu(Array a, int n) {
    int i;
    Array aInv, devA, devAInv;
    hipblasHandle_t handle;

    aInv = (Array)malloc(ArraySize);
    ensure(aInv, "could not allocate 0x%lX bytes of memory for matrix inverse", ArraySize);

    memset(aInv, 0, ArraySize);
    for (i = 0; i < n; ++i) { aInv[i*n + i] = 1.f; }

    cublasErrchk( hipblasCreate(&handle) );

    gpuErrchk( hipMalloc(&devA, ArraySize) );
    gpuErrchk( hipMalloc(&devAInv, ArraySize) );

    gpuErrchk( hipMemcpy(devA, a, ArraySize, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(devAInv, aInv, ArraySize, hipMemcpyHostToDevice) );

    /* Invert the matrix */
    invert(handle, devA, devAInv);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    /* Display the result */
    gpuErrchk( hipMemcpy(a, devAInv, ArraySize, hipMemcpyDeviceToHost) );

    /* Cleanup the mess */
    gpuErrchk( hipFree(devA) );
    gpuErrchk( hipFree(devAInv) );
    cublasErrchk( hipblasDestroy(handle) );
}

// int main(int argc, char *argv[]) {
//  Array a, a_inv;
//  Array dev_a, dev_a_inv;
//  hipblasHandle_t handle;

//  /* Pre-processing steps */
//  if(!(a = (Array)malloc(ArraySize))) {
//      perror("");
//      return errno;
//  }
//  if(!(a_inv = (Array)malloc(ArraySize))) {
//      perror("");
//      return errno;
//  }
//  hipMalloc(&dev_a, ArraySize);
//  hipMalloc(&dev_a_inv, ArraySize);
//  hipblasCreate(&handle);

//  /* Input column major matrix */
//  for(int i = 0; i < N; i++)
//      for(int j = 0; j < N; j++) {
//          scanf("%f", &a[i * N + j]);
//          if(i == j)
//              a_inv[i * N + j] = 1;
//          else
//              a_inv[i * N + j] = 0;
//      }
//  hipMemcpy(dev_a, a, ArraySize, hipMemcpyHostToDevice);
//  hipMemcpy(dev_a_inv, a_inv, ArraySize, hipMemcpyHostToDevice);

//  /* Invert the matrix */
//  invert(handle, dev_a, dev_a_inv);

//  /* Display the result */
//  hipMemcpy(a, dev_a, ArraySize, hipMemcpyDeviceToHost);
//  hipMemcpy(a_inv, dev_a_inv, ArraySize, hipMemcpyDeviceToHost);
//  printf("Inverse is:\n");
//  //printMatrix(a);
//  printMatrix(a_inv);

//  /* Cleanup the mess */
//  free(a);
//  free(a_inv);
//  hipFree(dev_a);
//  hipFree(dev_a_inv);
//  hipblasDestroy(handle);

//  return 0;
// }
