#include "hip/hip_runtime.h"
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../../include/types.h"
#include "../../include/helper_cpu.h"
#include "../../include/helper_gpu.h"
#include "../../include/inverse_cpu.h"
#include "../../include/inverse_gpu.h"

#define SWAP(x, y, z)	((z) = (x),(x) = (y),(y) = (z))

__global__
void pivotRow(Array *a, Array *a_inv, int n, int col) {
	__shared__ int row;

	if(a[blockIdx.x][col * n + col] != 0)
		return;

	// You can not add cublas error check here. Raises error
	hipblasHandle_t handle;
	int pivot;

	hipblasCreate(&handle);
	hipblasIsamax(handle,
		n - col,						// Number of elements to be searched
		a[blockIdx.x] + (col * n) + col,// Starting position
		1,								// Increment in words (NOT BYTES)
		&pivot);						// Maximum element in the col
	row = pivot - 1 + col;

	hipblasSswap(handle, n, a[blockIdx.x] + col, n, a[blockIdx.x] + row, n);
	hipblasSswap(handle, n, a_inv[blockIdx.x] + col, n, a_inv[blockIdx.x] + row, n);
	hipblasDestroy(handle);
}

__global__
void normalizeRow(Array *a, Array *a_inv, int n, int row) {
	__shared__ DataType scalar;

	if(threadIdx.x == 0)
		scalar = 1 / a[blockIdx.x][row * n + row];
	__syncthreads();

	a[blockIdx.x][threadIdx.x * n + row] *= scalar;
	a_inv[blockIdx.x][threadIdx.x * n + row] *= scalar;
}

__global__
void transform_matrix(Array *a, Array *a_inv, int n, int row) {
	extern __shared__ DataType shared[];

	DataType *scalars = &shared[0];
	DataType *currRowA = &shared[n];
	DataType *currRowI = &shared[2 * n];

	// store the scalars corresponding to the column 'row'
	scalars[threadIdx.x] = a[blockIdx.x][row * n + threadIdx.x];
	currRowA[threadIdx.x] = a[blockIdx.x][threadIdx.x * n + row];
	currRowI[threadIdx.x] = a_inv[blockIdx.x][threadIdx.x * n + row];
	__syncthreads();

	// no need to transform 'row'th row
	if(threadIdx.x == row)
		return;

	// Each thread transforms row
	for(int i = 0; i < n; i++) {
		a[blockIdx.x][i * n + threadIdx.x] -= (scalars[threadIdx.x] * currRowA[i]);
		a_inv[blockIdx.x][i * n + threadIdx.x] -= (scalars[threadIdx.x] * currRowI[i]);
	}
}

void invert(hipblasHandle_t &handle, int n, Array *a, Array *a_inv, int batchSize) {
	for(int i = 0; i < n; i++) {
		// Pivot the matrix
		pivotRow<<<batchSize, n>>>(a, a_inv, n, i);

		// Make column entry to be one
		normalizeRow<<<batchSize, n>>>(a, a_inv, n, i);

		// number of threads equals number of rows
		transform_matrix<<<batchSize, n, 3*n*sizeof(DataType)>>>(a, a_inv, n, i);
	}
}

void inverse_gauss_batched_device(hipblasHandle_t handle, int n, Array devAs, Array devAInvs, int batchSize);

extern "C" void inverse_gauss_batched_gpu(
		hipblasHandle_t handle,
		int n,
		Array As,
		Array aInvs,
		int batchSize) {

	int k, i;
	Array *devAs;
	size_t pitchAs;
	Array *devAInvs;
	size_t pitchAInvs;

	const size_t ArraySize = sizeof(DataType) * n * n;

	gpuErrchk( hipHostAlloc((void**)&devAs, sizeof(Array)*batchSize, hipHostMallocDefault) );
	gpuErrchk( hipHostAlloc((void**)&devAInvs, sizeof(Array)*batchSize, hipHostMallocDefault) );

	gpuErrchk( batchedCudaMalloc(devAs, &pitchAs, ArraySize, batchSize) );
	gpuErrchk( batchedCudaMalloc(devAInvs, &pitchAInvs, ArraySize, batchSize) );

    memset(aInvs, 0, batchSize*ArraySize);

	for (k = 0; k < batchSize; ++k) {
	    for (i = 0; i < n; ++i) {
	    	aInvs[k*n*n + i*n + i] = 1.f;
    	}
	}

	gpuErrchk( hipMemcpy2D(devAs[0], pitchAs, As, ArraySize, ArraySize, batchSize,
				hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy2D(devAInvs[0], pitchAInvs, aInvs, ArraySize, ArraySize, batchSize,
				hipMemcpyHostToDevice) );

	// Calculate Minv = Madd^-1, store result in Bs
	invert(handle, n, devAs, devAInvs, batchSize);
	// devAs: As
	// devAs: Minv
	// devAInvs: Madd

	gpuErrchk( hipMemcpy2D(aInvs, ArraySize, devAInvs[0], pitchAInvs, ArraySize, batchSize,
				hipMemcpyDeviceToHost) );
	gpuErrchk( hipFree((void*)devAs[0]) );
	gpuErrchk( hipFree((void*)devAInvs[0]) );
	gpuErrchk( hipHostFree((void*)devAs) );
	gpuErrchk( hipHostFree((void*)devAInvs) );
}

// int main(int argc, char *argv[]) {
// 	hipblasHandle_t handle;
// 	int numMatrices, n;
// 	Array a, a_inv;

// 	cublasErrchk( hipblasCreate(&handle) );

// 	readMatricesFile(argv[1], &numMatrices, &n, &n, &a);
// 	a_inv = (Array) malloc(sizeof(DataType) * numMatrices * n * n);
// 	printMatrixList(a, n, numMatrices);
// 	for(int i = 0; i < numMatrices; i++)
// 		for(int j = 0; j < n; j++)
// 			for(int k = 0; k < n; k++)
// 				if(j == k)
// 					a_inv[i * n * n + j * n + k] = 1;
// 				else
// 					a_inv[i * n * n + j * n + k] = 0;
// 	batchedInverse(handle, n, a, a_inv, numMatrices);
// 	printMatrixList(a_inv, n, numMatrices);

// 	gpuErrchk( hipPeekAtLastError() );
// 	gpuErrchk( hipDeviceSynchronize() );

// 	return 0;
// }
