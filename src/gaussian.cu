#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define ELEMENT_TYPE float

static const ELEMENT_TYPE ELEMENT_ZERO = ELEMENT_TYPE(0);
static const ELEMENT_TYPE ELEMENT_ONE = ELEMENT_TYPE(1);

typedef ELEMENT_TYPE *Array;

// Memory model
// ************
//
// All matrices are allocated on a continouse portion of memory.
//
// PRO
// + Easy to transfer to and from the GPU as it is one memcpy
// + Allocation goes a lot quicker (needs profiling) since memory can be allocated in one portion
//   and we do not need to loop to allocate memory on the GPU
//
// CON
// - We need to generate the array of pointers to use the xxxBatched APIs
// - Some memory access in the kernels will be missaligned which can hamper performance (needs profiling)
//      > Seems like this can be avoided with cudaMallocPitch and cudaMemcpy2D

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s:%d\n", hipGetErrorString(code), file, line);
        if (abort) { exit(code); }
    }
}

// Simple 8-bit bit reversal Compute test

#define N 256

__global__ void bitreverse(unsigned int *data)
{
    unsigned int *idata = data;

    unsigned int x = idata[threadIdx.x];

    x = ((0xf0f0f0f0 & x) >> 4) | ((0x0f0f0f0f & x) << 4);
    x = ((0xcccccccc & x) >> 2) | ((0x33333333 & x) << 2);
    x = ((0xaaaaaaaa & x) >> 1) | ((0x55555555 & x) << 1);

    idata[threadIdx.x] = x;
}

// Allocates one continous array of memory of size arraySize*batchSize and writes the
// pointers of all subarrays into the array of pointers located at devArrayPtr.
static hipError_t batchedCudaMalloc(Array* devArrayPtr, size_t *pitch, size_t arraySize, int batchSize) {
    char *devPtr;

    hipError_t result = hipMallocPitch((void**)&devPtr, pitch, arraySize, batchSize);

    if (hipSuccess != result) {
        return result;
    }
    
    for (int i = 0; i < batchSize; ++i) {
        devArrayPtr[i] = (Array)devPtr;
        devPtr += *pitch;
    }

    return hipSuccess;
}

// Adds all matrices in devLeft to their corresponding matrix in devRight.
// The data inside devRight is modified, devLeft is left untouched.
// Both devLeft and devRight are expected to be already allocated on the GPU.
// defRight += devLeft
static void batchedAdd(
    hipblasHandle_t handle,
    int n,
    const ELEMENT_TYPE *alpha,
    const Array devLeft[],
    Array devRight[],
    int batchSize) {
    // TODO: implement addition. Can also be done on the CPU but then we
    // need to do it before transferring the data to the GPU.
    // SEE: http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-axpy
    // SEE: http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-geam
}

// Inverts all matrices in devMatrices and stores the result in devInvMatrices.
// devMatrices and devInvMatrices are already allocated on the GPU. However,
// maybe one of the two methods for inversion does not need a workspace the size
// of the input. In that case this function signature has to change!
// defInvMatrices = devMatrices^{-1}
static void batchedInverse(
    hipblasHandle_t handle,
    int n,
    const Array devMatrices[],
    Array devInvMatrices[],
    int batchSize) {
    // TODO: implement matrix inversion. Please see how you want to dispatch to the corresponding inversion algorithm.
    // SEE: http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-getrfbatched
    // SEE: http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-getribatched
}

// Multiplies all matrices in devLeft to their corresponding matrix in devRight.
// The data inside devLeft and devRight is untouched, devResult is modified.
// devLeft, devRight and devResult are expected to be already allocated on the GPU.
// m = number of rows of transa(devLeft) and devResult
// n = number of columns of transb(devRight) and devResult
// k = number of columns of transa(devLeft) and rows of transb(devRight)
// devResult = alpha*transa(devLeft)*transb(devRight) + beta*devResult
static void batchedMul(
    hipblasHandle_t handle,
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const ELEMENT_TYPE *alpha,
    const Array devLeft[],
    const Array devReight[],
    const ELEMENT_TYPE *beta,
    Array devResult[],
    int batchSize) {
    // TODO: implement matrix multiplication.
    // SEE: http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemmbatched
}

// Calculates the mean of the matrix set {A, B, C, D}.
// Mean = A*(B+C)^{-1}*D
// As       batchSize x n x 1
// Bs       batchSize x n x n
// Cs       batchSize x n x n
// Ds       batchSize x n x 1
// Means    batchSize x n x 1
// Means is assumed to be already allocated.
static void calcluate_mean(
    hipblasHandle_t handle,
    int n,
    Array As,
    Array Bs,
    Array Cs,
    Array Ds,
    Array Means,
    int batchSize) {

    Array *devBs;
    size_t pitchBs;
    Array *devCs;
    size_t pitchCs;
    Array *devDs;
    size_t pitchDs;

    const size_t sizeOfMatrixA = sizeof(ELEMENT_TYPE)*n;
    const size_t sizeOfMatrixB = sizeof(ELEMENT_TYPE)*n*n;
    const size_t sizeOfMatrixC = sizeof(ELEMENT_TYPE)*n*n;
    const size_t sizeOfMatrixD = sizeof(ELEMENT_TYPE)*n;
    const size_t sizeOfResult = sizeof(ELEMENT_TYPE);

    gpuErrchk( hipHostAlloc((void**)&devBs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devCs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devDs, sizeof(Array)*batchSize, hipHostMallocDefault) );

    // Allocate and copy Bs, Cs and Ds to the GPU
    gpuErrchk( batchedCudaMalloc(devBs, &pitchBs, sizeOfMatrixB, batchSize) );
    gpuErrchk( batchedCudaMalloc(devCs, &pitchCs, sizeOfMatrixC, batchSize) );
    gpuErrchk( batchedCudaMalloc(devDs, &pitchDs, sizeOfMatrixD, batchSize) );

    gpuErrchk( hipMemcpy2D(devBs, pitchBs, Bs, sizeOfMatrixB, sizeOfMatrixB, batchSize,
               hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy2D(devCs, pitchCs, Cs, sizeOfMatrixC, sizeOfMatrixC, batchSize,
               hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy2D(devDs, pitchDs, Ds, sizeOfMatrixD, sizeOfMatrixD, batchSize,
               hipMemcpyHostToDevice) );

    // Calculate Madd = B + C for every matrix, store result in Cs
    batchedAdd(handle, n, &ELEMENT_ONE, devBs, devCs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devBs: Bs
    // devCs: Madd
    // devDs: Ds

    // Calculate Minv = Madd^-1, store result in Bs
    batchedInverse(handle, n, devCs, devBs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devBs: Minv
    // devCs: Madd
    // devDs: Ds

    // Calculate Mmul = Minv * Ds, store result in Cs
    batchedMul(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, 1, &ELEMENT_ONE, devBs, devDs, &ELEMENT_ZERO, devCs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devBs: Minv
    // devCs: Mmul
    // devDs: Ds

    // Load As into GPU memory overwriting devDs.
    gpuErrchk( hipMemcpy2D(devDs, pitchDs, As, sizeOfMatrixA, sizeOfMatrixA, batchSize,
               hipMemcpyHostToDevice) );
    // devBs: Minv
    // devCs: Mmul
    // devDs: As

    // Calculate Mmean = AT * Mmul + (whatever is in Bs), store result in Bs
    batchedMul(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, n, n, &ELEMENT_ONE, devCs, devDs, &ELEMENT_ZERO, devBs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devBs: Mmean
    // devCs: Mmul
    // devDs: As

    // Fetch result from GPU and free used memory.
    gpuErrchk( hipMemcpy2D(devBs, pitchBs, Means, sizeOfResult, sizeOfResult, batchSize,
               hipMemcpyHostToDevice) );

    gpuErrchk( hipFree((void*)devBs[0]) );
    gpuErrchk( hipFree((void*)devCs[0]) );
    gpuErrchk( hipFree((void*)devDs[0]) );

    gpuErrchk( hipHostFree((void*)devBs) );
    gpuErrchk( hipHostFree((void*)devCs) );
    gpuErrchk( hipHostFree((void*)devDs) );
}

// Calculates the variance of the matrix set {A, B, C, D, E}.
// Var = E-AT*(B+C)^{-1}*A
// As       batchSize x n x 1
// Bs       batchSize x n x n
// Cs       batchSize x n x n
// Ds       batchSize x n x 1
// Means    batchSize x n x 1
// Means is assumed to be already allocated.
static void calcluate_variance(
    hipblasHandle_t handle,
    int n,
    Array As,
    Array Bs,
    Array Cs,
    Array Ds,
    Array Es,
    Array Variances,
    int batchSize) {

    Array *devAs;
    size_t pitchAs;
    Array *devBs;
    size_t pitchBs;
    Array *devCs;
    size_t pitchCs;

    const size_t sizeOfMatrixA = sizeof(ELEMENT_TYPE)*n;
    const size_t sizeOfMatrixB = sizeof(ELEMENT_TYPE)*n*n;
    const size_t sizeOfMatrixC = sizeof(ELEMENT_TYPE)*n*n;
    const size_t sizeOfMatrixE = sizeof(ELEMENT_TYPE);

    gpuErrchk( hipHostAlloc((void**)&devAs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devBs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devCs, sizeof(Array)*batchSize, hipHostMallocDefault) );

    // Allocate and copy Bs, Cs and As to the GPU
    gpuErrchk( batchedCudaMalloc(devAs, &pitchAs, sizeOfMatrixA, batchSize) );
    gpuErrchk( batchedCudaMalloc(devBs, &pitchBs, sizeOfMatrixB, batchSize) );
    gpuErrchk( batchedCudaMalloc(devCs, &pitchCs, sizeOfMatrixC, batchSize) );

    gpuErrchk( hipMemcpy2D(devAs, pitchAs, As, sizeOfMatrixA, sizeOfMatrixA, batchSize,
               hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy2D(devBs, pitchBs, Bs, sizeOfMatrixB, sizeOfMatrixB, batchSize,
               hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy2D(devCs, pitchCs, Cs, sizeOfMatrixC, sizeOfMatrixC, batchSize,
               hipMemcpyHostToDevice) );

    // Calculate Madd = B + C for every matrix, store result in Cs
    batchedAdd(handle, n, &ELEMENT_ONE, devBs, devCs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devAs: As
    // devBs: Bs
    // devCs: Madd

    // Calculate Minv = Madd^-1, store result in Bs
    batchedInverse(handle, n, devCs, devBs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devAs: As
    // devBs: Minv
    // devCs: Madd

    // Calculate Mmul = Minv * A + (whatever is in Cs), store result in Cs
    batchedMul(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, 1, &ELEMENT_ONE, devBs, devAs, &ELEMENT_ZERO, devCs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devAs: As
    // devBs: Minv
    // devCs: Mmul

    // Calculate Mmul2 = AT * Mmul + (whatever is in Bs), store result in Bs
    batchedMul(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, n, n, &ELEMENT_ONE, devCs, devAs, &ELEMENT_ZERO, devBs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devAs: As
    // devBs: Mmul2
    // devCs: Mmul

    // Load Es to the GPU overwriting As
    gpuErrchk( hipMemcpy2D(devAs, pitchAs, Es, sizeOfMatrixE, sizeOfMatrixE, batchSize,
               hipMemcpyHostToDevice) );

    const ELEMENT_TYPE minusOne = ELEMENT_TYPE(-1);
    batchedAdd(handle, n, &minusOne, devBs, devAs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Fetch result from GPU and free used memory.
    gpuErrchk( hipMemcpy2D(devAs, pitchAs, Variances, sizeOfMatrixE, sizeOfMatrixE, batchSize,
               hipMemcpyHostToDevice) );

    gpuErrchk( hipFree((void*)devAs[0]) );
    gpuErrchk( hipFree((void*)devBs[0]) );
    gpuErrchk( hipFree((void*)devCs[0]) );

    gpuErrchk( hipHostFree((void*)devAs) );
    gpuErrchk( hipHostFree((void*)devBs) );
    gpuErrchk( hipHostFree((void*)devCs) );
}

int main(void)
{
    unsigned int *d = NULL; int i;
    unsigned int idata[N], odata[N];

    for (i = 0; i < N; i++)
         idata[i] = (unsigned int)i;

    gpuErrchk( hipMalloc((void**)&d, sizeof(int)*N)) ;
    gpuErrchk( hipMemcpy(d, idata, sizeof(int)*N,
               hipMemcpyHostToDevice) );

    bitreverse<<<1, N>>>(d);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipMemcpy(odata, d, sizeof(int)*N,
               hipMemcpyDeviceToHost) );

    for (i = 0; i < N; i++)
        printf("%u -> %u\n", idata[i], odata[i]);

    gpuErrchk( hipFree((void*)d) );
    return 0;
}
