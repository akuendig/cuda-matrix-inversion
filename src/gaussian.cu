#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define MAX_MATRIX_BYTE_READ 67108864

#define ELEMENT_TYPE float

static const ELEMENT_TYPE ELEMENT_ZERO = ELEMENT_TYPE(0);
static const ELEMENT_TYPE ELEMENT_ONE = ELEMENT_TYPE(1);

typedef ELEMENT_TYPE *Array;

// Memory model
// ************
//
// All matrices are allocated on a continouse portion of memory.
//
// PRO
// + Easy to transfer to and from the GPU as it is one memcpy
// + Allocation goes a lot quicker (needs profiling) since memory can be allocated in one portion
//   and we do not need to loop to allocate memory on the GPU
//
// CON
// - We need to generate the array of pointers to use the xxxBatched APIs
// - Some memory access in the kernels will be missaligned which can hamper performance (needs profiling)
//      > Seems like this can be avoided with hipMallocPitch and hipMemcpy2D

#define fail(...) \
  fprintf(stderr, "%s:%d\t", __FILE__, __LINE__); \
  fprintf(stderr, __VA_ARGS__); \
  fprintf(stderr, "\r\n"); \
  exit(EXIT_FAILURE);

#define ensure(condition, ...) \
  do { \
    if (! (condition)) { \
      fprintf(stderr, "ENSURE FAILED %s:%d\r\n", __FILE__, __LINE__); \
      fprintf(stderr, __VA_ARGS__); \
      fprintf(stderr, "\r\n"); \
      if (errno) { perror("possible reason for failure from ERRNO"); } \
      exit(EXIT_FAILURE); \
    } \
  } while(0)

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s:%d\n", hipGetErrorString(code), file, line);
        hipDeviceReset();
        if (abort) { exit(code); }
    }
}

// Simple 8-bit bit reversal Compute test

#define N 256

__global__ void bitreverse(unsigned int *data)
{
    unsigned int *idata = data;

    unsigned int x = idata[threadIdx.x];

    x = ((0xf0f0f0f0 & x) >> 4) | ((0x0f0f0f0f & x) << 4);
    x = ((0xcccccccc & x) >> 2) | ((0x33333333 & x) << 2);
    x = ((0xaaaaaaaa & x) >> 1) | ((0x55555555 & x) << 1);

    idata[threadIdx.x] = x;
}

// Allocates one continous array of memory of size arraySize*batchSize and writes the
// pointers of all subarrays into the array of pointers located at devArrayPtr.
static hipError_t batchedCudaMalloc(Array* devArrayPtr, size_t *pitch, size_t arraySize, int batchSize) {
    char *devPtr;

    hipError_t result = hipMallocPitch((void**)&devPtr, pitch, arraySize, batchSize);

    if (hipSuccess != result) {
        return result;
    }

    for (int i = 0; i < batchSize; ++i) {
        devArrayPtr[i] = (Array)devPtr;
        devPtr += *pitch;
    }

    return hipSuccess;
}

// Adds all matrices in devLeft to their corresponding matrix in devRight.
// The data inside devRight is modified, devLeft is left untouched.
// Both devLeft and devRight are expected to be already allocated on the GPU.
// defRight += devLeft
static void batchedAdd(
    hipblasHandle_t handle,
    int n,
    const ELEMENT_TYPE *alpha,
    const Array devLeft[],
    Array devRight[],
    int batchSize) {
    // TODO: implement addition. Can also be done on the CPU but then we
    // need to do it before transferring the data to the GPU.
    // SEE: http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-axpy
    // SEE: http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-geam
}

// Inverts all matrices in devMatrices and stores the result in devInvMatrices.
// devMatrices and devInvMatrices are already allocated on the GPU. However,
// maybe one of the two methods for inversion does not need a workspace the size
// of the input. In that case this function signature has to change!
// defInvMatrices = devMatrices^{-1}
static void batchedInverse(
    hipblasHandle_t handle,
    int n,
    const Array devMatrices[],
    Array devInvMatrices[],
    int batchSize) {
    // TODO: implement matrix inversion. Please see how you want to dispatch to the corresponding inversion algorithm.
    // SEE: http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-getrfbatched
    // SEE: http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-getribatched
}

// Multiplies all matrices in devLeft to their corresponding matrix in devRight.
// The data inside devLeft and devRight is untouched, devResult is modified.
// devLeft, devRight and devResult are expected to be already allocated on the GPU.
// m = number of rows of transa(devLeft) and devResult
// n = number of columns of transb(devRight) and devResult
// k = number of columns of transa(devLeft) and rows of transb(devRight)
// devResult = alpha*transa(devLeft)*transb(devRight) + beta*devResult
static void batchedMul(
    hipblasHandle_t handle,
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const ELEMENT_TYPE *alpha,
    const Array devLeft[],
    const Array devReight[],
    const ELEMENT_TYPE *beta,
    Array devResult[],
    int batchSize) {
    // TODO: implement matrix multiplication.
    // SEE: http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemmbatched
}

// Calculates the mean of the matrix set {A, B, C, D}.
// Mean = A*(B+C)^{-1}*D
// As       batchSize x n x 1
// Bs       batchSize x n x n
// Cs       batchSize x n x n
// Ds       batchSize x n x 1
// Means    batchSize x n x 1
// Means is assumed to be already allocated.
static void calcluateMean(
    hipblasHandle_t handle,
    int n,
    Array As,
    Array Bs,
    Array Cs,
    Array Ds,
    Array Means,
    int batchSize) {

    Array *devBs;
    size_t pitchBs;
    Array *devCs;
    size_t pitchCs;
    Array *devDs;
    size_t pitchDs;

    const size_t sizeOfMatrixA = sizeof(ELEMENT_TYPE)*n;
    const size_t sizeOfMatrixB = sizeof(ELEMENT_TYPE)*n*n;
    const size_t sizeOfMatrixC = sizeof(ELEMENT_TYPE)*n*n;
    const size_t sizeOfMatrixD = sizeof(ELEMENT_TYPE)*n;
    const size_t sizeOfResult = sizeof(ELEMENT_TYPE);

    gpuErrchk( hipHostAlloc((void**)&devBs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devCs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devDs, sizeof(Array)*batchSize, hipHostMallocDefault) );

    // Allocate and copy Bs, Cs and Ds to the GPU
    gpuErrchk( batchedCudaMalloc(devBs, &pitchBs, sizeOfMatrixB, batchSize) );
    gpuErrchk( batchedCudaMalloc(devCs, &pitchCs, sizeOfMatrixC, batchSize) );
    gpuErrchk( batchedCudaMalloc(devDs, &pitchDs, sizeOfMatrixD, batchSize) );

    gpuErrchk( hipMemcpy2D(devBs, pitchBs, Bs, sizeOfMatrixB, sizeOfMatrixB, batchSize,
               hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy2D(devCs, pitchCs, Cs, sizeOfMatrixC, sizeOfMatrixC, batchSize,
               hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy2D(devDs, pitchDs, Ds, sizeOfMatrixD, sizeOfMatrixD, batchSize,
               hipMemcpyHostToDevice) );

    // Calculate Madd = B + C for every matrix, store result in Cs
    batchedAdd(handle, n, &ELEMENT_ONE, devBs, devCs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devBs: Bs
    // devCs: Madd
    // devDs: Ds

    // Calculate Minv = Madd^-1, store result in Bs
    batchedInverse(handle, n, devCs, devBs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devBs: Minv
    // devCs: Madd
    // devDs: Ds

    // Calculate Mmul = Minv * Ds, store result in Cs
    batchedMul(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, 1, &ELEMENT_ONE, devBs, devDs, &ELEMENT_ZERO, devCs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devBs: Minv
    // devCs: Mmul
    // devDs: Ds

    // Load As into GPU memory overwriting devDs.
    gpuErrchk( hipMemcpy2D(devDs, pitchDs, As, sizeOfMatrixA, sizeOfMatrixA, batchSize,
               hipMemcpyHostToDevice) );
    // devBs: Minv
    // devCs: Mmul
    // devDs: As

    // Calculate Mmean = AT * Mmul + (whatever is in Bs), store result in Bs
    batchedMul(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, n, n, &ELEMENT_ONE, devCs, devDs, &ELEMENT_ZERO, devBs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devBs: Mmean
    // devCs: Mmul
    // devDs: As

    // Fetch result from GPU and free used memory.
    gpuErrchk( hipMemcpy2D(devBs, pitchBs, Means, sizeOfResult, sizeOfResult, batchSize,
               hipMemcpyHostToDevice) );

    gpuErrchk( hipFree((void*)devBs[0]) );
    gpuErrchk( hipFree((void*)devCs[0]) );
    gpuErrchk( hipFree((void*)devDs[0]) );

    gpuErrchk( hipHostFree((void*)devBs) );
    gpuErrchk( hipHostFree((void*)devCs) );
    gpuErrchk( hipHostFree((void*)devDs) );
}

// Calculates the variance of the matrix set {A, B, C, D, E}.
// Var = E-AT*(B+C)^{-1}*A
// As       batchSize x n x 1
// Bs       batchSize x n x n
// Cs       batchSize x n x n
// Ds       batchSize x n x 1
// Means    batchSize x n x 1
// Means is assumed to be already allocated.
static void calcluateVariance(
    hipblasHandle_t handle,
    int n,
    Array As,
    Array Bs,
    Array Cs,
    Array Ds,
    Array Es,
    Array Variances,
    int batchSize) {

    Array *devAs;
    size_t pitchAs;
    Array *devBs;
    size_t pitchBs;
    Array *devCs;
    size_t pitchCs;

    const size_t sizeOfMatrixA = sizeof(ELEMENT_TYPE)*n;
    const size_t sizeOfMatrixB = sizeof(ELEMENT_TYPE)*n*n;
    const size_t sizeOfMatrixC = sizeof(ELEMENT_TYPE)*n*n;
    const size_t sizeOfMatrixE = sizeof(ELEMENT_TYPE);

    gpuErrchk( hipHostAlloc((void**)&devAs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devBs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devCs, sizeof(Array)*batchSize, hipHostMallocDefault) );

    // Allocate and copy Bs, Cs and As to the GPU
    gpuErrchk( batchedCudaMalloc(devAs, &pitchAs, sizeOfMatrixA, batchSize) );
    gpuErrchk( batchedCudaMalloc(devBs, &pitchBs, sizeOfMatrixB, batchSize) );
    gpuErrchk( batchedCudaMalloc(devCs, &pitchCs, sizeOfMatrixC, batchSize) );

    gpuErrchk( hipMemcpy2D(devAs, pitchAs, As, sizeOfMatrixA, sizeOfMatrixA, batchSize,
               hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy2D(devBs, pitchBs, Bs, sizeOfMatrixB, sizeOfMatrixB, batchSize,
               hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy2D(devCs, pitchCs, Cs, sizeOfMatrixC, sizeOfMatrixC, batchSize,
               hipMemcpyHostToDevice) );

    // Calculate Madd = B + C for every matrix, store result in Cs
    batchedAdd(handle, n, &ELEMENT_ONE, devBs, devCs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devAs: As
    // devBs: Bs
    // devCs: Madd

    // Calculate Minv = Madd^-1, store result in Bs
    batchedInverse(handle, n, devCs, devBs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devAs: As
    // devBs: Minv
    // devCs: Madd

    // Calculate Mmul = Minv * A + (whatever is in Cs), store result in Cs
    batchedMul(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, 1, &ELEMENT_ONE, devBs, devAs, &ELEMENT_ZERO, devCs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devAs: As
    // devBs: Minv
    // devCs: Mmul

    // Calculate Mmul2 = AT * Mmul + (whatever is in Bs), store result in Bs
    batchedMul(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, n, n, &ELEMENT_ONE, devCs, devAs, &ELEMENT_ZERO, devBs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // devAs: As
    // devBs: Mmul2
    // devCs: Mmul

    // Load Es to the GPU overwriting As
    gpuErrchk( hipMemcpy2D(devAs, pitchAs, Es, sizeOfMatrixE, sizeOfMatrixE, batchSize,
               hipMemcpyHostToDevice) );

    const ELEMENT_TYPE minusOne = ELEMENT_TYPE(-1);
    batchedAdd(handle, n, &minusOne, devBs, devAs, batchSize);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Fetch result from GPU and free used memory.
    gpuErrchk( hipMemcpy2D(devAs, pitchAs, Variances, sizeOfMatrixE, sizeOfMatrixE, batchSize,
               hipMemcpyHostToDevice) );

    gpuErrchk( hipFree((void*)devAs[0]) );
    gpuErrchk( hipFree((void*)devBs[0]) );
    gpuErrchk( hipFree((void*)devCs[0]) );

    gpuErrchk( hipHostFree((void*)devAs) );
    gpuErrchk( hipHostFree((void*)devBs) );
    gpuErrchk( hipHostFree((void*)devCs) );
}

void readMatricesFile(const char *path, int *numMatrices, int *m, int *n, Array *matrices) {
    int ret;
    int _numMatrics, _m, _n;

    FILE* fp = fopen(path, "r");
    ensure(fp, "could not open matrix file %s", path);

    ret = fscanf(fp, "%d %d %d", &_numMatrices, &_m, &_n);
    ensure(3 == ret, "could not read number of matrices from file %s", path);

    *numMatrices = _numMatrices;
    *m = _m;
    *n = _n;

    size_t arraySize = sizeof(ELEMENT_TYPE) * (_numMatrices) * (_m) * (_n);
    ensure(arraySize <= MAX_MATRIX_BYTE_READ, "cannot read file %s because "
        "the allocated array would be bigger than 0x%X bytes", path, arraySize);

    *matrices = malloc(arraySize);
    ensure(*matrices, "could not allocate 0x%X bytes of memory for file %s", arraySize, path);

    Array currentElement = *matrices;

    for (int k = 0; k < _numMatrices; ++k) {
        for (int i = 0; i < _m; ++i) {
            for (int j = 0; j < _n; ++j, ++currentElement) {
                ret = fscanf(fp, "%d", currentElement);
                ensure(ret, "could not read matrix from file %s, stuck at matrix %d element %d, %d", path, k, i, j);
            }
        }
    }

    fclose(fp);
}

void readMeanTest(const char *directory, int *numMatrices, int *n,
        Array *a, Array *b, Array *c, Array *d) {
    char filePath[1024];

    int numMatricesA, numMatricesB, numMatricesC, numMatricesD;
    int mA, mB, mC, mD;
    int nA, nB, nC, nD;
    Array a, b, c, d;

    snprintf(filePath, 1024, "%s/a.mats", directory);
    readMatricesFile(filePath, &numMatricesA, &mA, &nA, &a);

    snprintf(filePath, 1024, "%s/b.mats", directory);
    readMatricesFile(filePath, &numMatricesB, &mB, &nB, &b);

    snprintf(filePath, 1024, "%s/c.mats", directory);
    readMatricesFile(filePath, &numMatricesC, &mC, &nC, &c);

    snprintf(filePath, 1024, "%s/d.mats", directory);
    readMatricesFile(filePath, &numMatricesD, &mD, &nD, &d);

    ensure(
        mA == mB && mB == mC && mC == mD &&
        nA == 1 && nB == mB && nC == mC && nD == 1,
        "test in directory %s invalid, dimensions not matching\r\n"
        "mA(%d) mB(%d) mC(%d) mD(%d)\r\n",
        "nA(%d) nB(%d) nC(%d) nD(%d)\r\n",
        mA, mB, mC, mD, nA, nB, nC, nD
    );
}

int main(void)
{
    hipblasHandle_t handle;

    int numMatrices, n;
    Array a, b, c, d;
    Array means;

    readMeanTest("tests/simpleMean", &numMatrices, &n, &a, &b, &c, *d);

    gpuErrchk( hipblasCreate(&handle) );
    gpuErrchk( hipMalloc((void**)&means, sizeof(ELEMENT_TYPE)*numMatrices)) ;


    calcluateMean(handle, n, a, b, c, d, means, numMatrices);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    for (int i = 0; i < numMatrices; i++) {
        printf("%f\r\n", means[i]);
    }

    gpuErrchk( hipFree((void*)means) );
    gpuErrchk( hipblasDestroy(&handle) );

    return 0;
}
