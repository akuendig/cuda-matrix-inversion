#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <errno.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../include/types.h"
#include "../include/helper_cpu.h"
#include "../include/helper_gpu.h"
#include "../include/inverse_cpu.h"
#include "../include/inverse_gpu.h"
#include "../include/timer.h"

#define MAX_THREADS_PER_BLOCK 256
#define MIN_OPS 64
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))
#define LBOUND(start,ops,threadId) (threadId * ops + start)
#define UBOUND(end, start,ops,threadId) MIN((threadId + 1) * ops + start, end)


#ifdef __MACH__
#include <mach/clock.h>
#include <mach/mach.h>
#endif

void current_utc_time(struct timespec *ts) {
 
#ifdef __MACH__ // OS X does not have clock_gettime, use clock_get_time
  clock_serv_t cclock;
  mach_timespec_t mts;
  host_get_clock_service(mach_host_self(), CALENDAR_CLOCK, &cclock);
  clock_get_time(cclock, &mts);
  mach_port_deallocate(mach_task_self(), cclock);
  ts->tv_sec = mts.tv_sec;
  ts->tv_nsec = mts.tv_nsec;
#else
  clock_gettime(CLOCK_REALTIME, ts);
#endif
 
}


__global__
void decompose_cholesky_kernel_device_ops(Array *aInv, int N, int ops) {
    int i, j, row;

    for (row = 0; row < N; row++) {


        if (threadIdx.x == 0) { // computing diagonal elements in the first thread

            aInv[blockIdx.x][row * N + row] = sqrt(aInv[blockIdx.x][row * N + row]);
            //printf("[%d][%d] = sqrt([%d][%d])\n", row, row, row, row);

            for (j = (row + 1); j < N; j++) {
                aInv[blockIdx.x][row * N + j] /= aInv[blockIdx.x][row * N + row]; // divide by diagonal elemnents
                //printf("[%d][%d] /= [%d][%d]\n", j, row, row, row);
            }
        }

        __syncthreads(); // all diagonal elemnents need to be computed

        for (i = LBOUND(row + 1, ops, threadIdx.x); i < UBOUND(N, row + 1, ops, threadIdx.x); i++) {
            for (j = i; j < N; j++) {
                aInv[blockIdx.x][i * N + j] -= aInv[blockIdx.x][row * N + i] * aInv[blockIdx.x][row * N + j];
                //printf("[%d][%d] -= [%d][%d] * [%d][%d]\n", j, i, i, row, j, row);
            }
        }

        __syncthreads(); // compute row by row
    }

    // set zeroes
    for (i = LBOUND(0, ops, threadIdx.x); i < UBOUND(N, 0, ops, threadIdx.x); i++) {
        for (j = 0; j < i; j++) {
            aInv[blockIdx.x][i * N + j] = 0;
        }
    }

}


extern "C" 
void decompose_cholesky_batched_device_ops(hipblasHandle_t handle, int N, Array *devAs, Array *devAInvs, int batchSize) {
    int ops = N; //MAX(N * N / MIN(N * N, MAX_THREADS_PER_BLOCK), N);
    int threads = N;
    decompose_cholesky_kernel_device_ops<<< batchSize, threads>>>(devAInvs, N, ops);
}

__global__
void inverse_upper_kernel_device_ops(Array *aInv, int N, int ops) {

    int i, j, row;

    for (row = 0; row < N; row++) {

        if (threadIdx.x == 0) { // re-compute diagonal elements in the first thread
            aInv[blockIdx.x][row * N + row] = 1.0 / aInv[blockIdx.x][row * N + row];
            //printf("[%d][%d] = 1.0 / [%d][%d] = %f\n", row, row, row, row, aInv[blockIdx.x][row * N + row]);

            for (j = (row + 1); j < N; j++) {
                //printf("[%d][%d] = -[%d][%d] * [%d][%d] / [%d][%d] = -%f * %f / %f", j, row, j, row, row, row, j, j, aInv[blockIdx.x][row * N + j], aInv[blockIdx.x][row * N + row], aInv[blockIdx.x][j * N + j]);
                aInv[blockIdx.x][row * N + j] = 0 - aInv[blockIdx.x][row * N + j] * aInv[blockIdx.x][row * N + row] / aInv[blockIdx.x][j * N + j];
                //printf("= %f\n", aInv[blockIdx.x][row * N + j]);
            }
        }

        __syncthreads(); // all diagonal elemnents need to be re-computed

        for (i = LBOUND(row + 2, ops, threadIdx.x); i < UBOUND(N, row + 2, ops, threadIdx.x); i++) {

            for (j = row + 1; j < i; j++) {
                aInv[blockIdx.x][row * N + i] -= aInv[blockIdx.x][j * N + i] * aInv[blockIdx.x][row * N + j] / aInv[blockIdx.x][i * N + i];
                //printf("[%d][%d] -= [%d][%d] * [%d][%d] / [%d][%d]\n", i, row, i, j, j, row, i, i);
            }

            __syncthreads();
        }

        __syncthreads(); // compute row by row
    }
}

extern "C" 
void inverse_upper_batched_device_ops(hipblasHandle_t handle, int N, Array *devAs, Array *devAInvs, int batchSize) {
    int ops = N; //MAX(N * N / MIN(N * N, MAX_THREADS_PER_BLOCK), N);
    int threads = N;
    inverse_upper_kernel_device_ops<<< batchSize, threads>>>(devAInvs, N, ops);
}


__global__
void multiply_upper_kernel_device_ops(Array *aInv, int N, int ops) {
    int i, j, row;

    for (row = 0; row < N; row++) {

        for (i = LBOUND(row, ops, threadIdx.x); i < UBOUND(N, row, ops, threadIdx.x); i++) {
            aInv[blockIdx.x][i * N + row] = aInv[blockIdx.x][i * N + i] * aInv[blockIdx.x][row * N + i];
            //printf("[%d][%d] = [%d][%d] * [%d][%d]\n", i, row, i, i, i, row);

            for (j = i + 1; j < N; j++) {
                // use upper matrix as buffer for multiplication
                aInv[blockIdx.x][i * N + row] += aInv[blockIdx.x][i * N + j] * aInv[blockIdx.x][row * N + j];
                //printf("[%d][%d] += [%d][%d] * [%d][%d]\n", i, row, j, i, j, row);
            }
        }

        __syncthreads(); // compute row by row
    }

    __syncthreads();

    // set back to lower matrix
    for (i = LBOUND(0, ops, threadIdx.x); i < UBOUND(N, 0, ops, threadIdx.x); i++) {
        for (j = 0; j < i; j++) {
            aInv[blockIdx.x][j * N + i] = aInv[blockIdx.x][i * N + j];
        }
    }
}

extern "C" 
void multiply_upper_batched_device_ops(hipblasHandle_t handle, int N, Array *devAs, Array *devAInvs, int batchSize) {
    int ops = N; //MAX(N * N / MIN(N * N, MAX_THREADS_PER_BLOCK), N);
    int threads = N;
    multiply_upper_kernel_device_ops<<< batchSize, threads>>>(devAInvs, N, ops);
}

extern "C" 
void inverse_cholesky_batched_device_ops(hipblasHandle_t handle, int N, Array *devAs, Array *devAInvs, int batchSize) {
    decompose_cholesky_batched_device_ops(handle, N, devAs, devAInvs, batchSize);
    inverse_upper_batched_device_ops(handle, N, devAs, devAInvs, batchSize);
    multiply_upper_batched_device_ops(handle, N, devAs, devAInvs, batchSize);
}

extern "C" 
void inverse_cholesky_batched_gpu_ops(hipblasHandle_t handle, int n, Array As, Array aInvs, int batchSize) {

    Array *devAs;
    size_t pitchAs;
    Array *devAInvs;
    size_t pitchAInvs;

#ifdef DETAILED_LOGGING
    TIMER_INIT(inverse_cholesky_batched_gpu_ops_mem_htod)
    TIMER_INIT(inverse_cholesky_batched_gpu_ops_ker)
    TIMER_INIT(inverse_cholesky_batched_gpu_ops_mem_dtoh)
#endif // DETAILED_LOGGING

    const size_t ArraySize = sizeof(DataType) * n * n;

    gpuErrchk( hipHostAlloc((void**)&devAs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devAInvs, sizeof(Array)*batchSize, hipHostMallocDefault) );

    gpuErrchk( batchedCudaMalloc(devAs, &pitchAs, ArraySize, batchSize) );
    gpuErrchk( batchedCudaMalloc(devAInvs, &pitchAInvs, ArraySize, batchSize) );

#ifdef DETAILED_LOGGING
    TIMER_START(inverse_cholesky_batched_gpu_ops_mem_htod)
#endif // DETAILED_LOGGING

    gpuErrchk( hipMemcpy2D(devAInvs[0], pitchAs, As, ArraySize, ArraySize, batchSize,
                hipMemcpyHostToDevice) );

#ifdef DETAILED_LOGGING
    TIMER_STOP(inverse_cholesky_batched_gpu_ops_mem_htod)
    TIMER_START(inverse_cholesky_batched_gpu_ops_ker)
#endif // DETAILED_LOGGING

    inverse_cholesky_batched_device_ops(handle, n, devAs, devAInvs, batchSize);

#ifdef DETAILED_LOGGING
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    TIMER_STOP(inverse_cholesky_batched_gpu_ops_ker)
    TIMER_START(inverse_cholesky_batched_gpu_ops_mem_dtoh)
#endif // DETAILED_LOGGING

    gpuErrchk( hipMemcpy2D(aInvs, ArraySize, devAInvs[0], pitchAInvs, ArraySize, batchSize,
                hipMemcpyDeviceToHost) );

#ifdef DETAILED_LOGGING
    TIMER_STOP(inverse_cholesky_batched_gpu_ops_mem_dtoh)

    TIMER_LOG(inverse_cholesky_batched_gpu_ops_mem_htod)
    TIMER_LOG(inverse_cholesky_batched_gpu_ops_ker)
    TIMER_LOG(inverse_cholesky_batched_gpu_ops_mem_dtoh)
#endif // DETAILED_LOGGING

    gpuErrchk( hipFree((void*)devAs[0]) );
    gpuErrchk( hipFree((void*)devAInvs[0]) );
    gpuErrchk( hipHostFree((void*)devAs) );
    gpuErrchk( hipHostFree((void*)devAInvs) );
}


__global__
void pivot_cholesky_kernel_device(Array *a, int N, int row) {

    //printf("%d\n", row);
    if (threadIdx.x == 0)
        a[blockIdx.x][row * N + row] = sqrt(a[blockIdx.x][row * N + row]); // update diagonal elemnets
    __syncthreads();

    //printf("[%d][%d] = sqrt([%d][%d])\n", row, row, row, row);

    int j = threadIdx.x + row + 1;
    if (j < N) {
        a[blockIdx.x][j * N + row] = 0;
        a[blockIdx.x][row * N + j] /= a[blockIdx.x][row * N + row]; // divide by diagonal elemnents
    }
}


__global__
void decompose_cholesky_kernel_device(Array *a, int N, int row) {

    int i = threadIdx.x + row + 1;

    for (int j = i; j < N; j++) {
        //printf("[%d][%d] -= [%d][%d] x [%d][%d] \n", j, i, i, row, j, row);
        a[blockIdx.x][i * N + j] = a[blockIdx.x][i * N + j] - a[blockIdx.x][row * N + i] * a[blockIdx.x][row * N + j];
    }
}

__global__
void inverse_upper_kernel_device(Array *a, Array *aInv, int N, int row) {
    int i = threadIdx.x;

    aInv[blockIdx.x][i * N + row] = 0 - a[blockIdx.x][i * N + row] *  aInv[blockIdx.x][i * N + i] / a[blockIdx.x][row * N + row]; 
    for (int j = i + 1; j < row; j++) {
        aInv[blockIdx.x][i * N + row] -= a[blockIdx.x][j * N + row] * aInv[blockIdx.x][i * N + j] / a[blockIdx.x][row * N + row];
    }

    if (row == i) {
        //printf("%d\n %f", row, a[blockIdx.x][row * N + row]);
        aInv[blockIdx.x][row * N + row] = 1.0 / a[blockIdx.x][row * N + row];
    }
}

__global__
void multiply_upper_kernel_device(Array *a, Array *aInv, int N, int row) {
    int i = threadIdx.x;

    for (int j = 0; j <= i; j++) {
            aInv[blockIdx.x][i * N + j] += a[blockIdx.x][i * N + row] *  a[blockIdx.x][j * N + row];
            aInv[blockIdx.x][j * N + i] = aInv[blockIdx.x][i * N + j];
            //printf("[%d][%d] += [%d][%d] * [%d][%d] %f\n", i, j, row, i, row, j, a[blockIdx.x][i * N + row] *  a[blockIdx.x][j * N + row]);
    }
}

__global__
void intialize_array(Array *a, int N) {
    for (int j = 0; j < N; j++) {
        a[blockIdx.x][threadIdx.x * N + j] = 0.0;
    }
}


extern "C" 
void inverse_cholesky_batched_device(hipblasHandle_t handle, int N, Array *devAs, Array *devAInvs, int batchSize) {
    Array *devATmp;
    size_t pitchATmp;

    gpuErrchk( hipHostAlloc((void**)&devATmp, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( batchedCudaMalloc(devATmp, &pitchATmp, sizeof(DataType) * N * N, batchSize) );


    // Set to zeroes
    intialize_array<<<batchSize, N>>>(devAInvs, N);

    for (int row = 0; row < N; row++) { // loop through each row
        
        // computing diagonal elements
        pivot_cholesky_kernel_device<<<batchSize, (N - row)>>>(devAs, N, row);
        gpuErrchk( hipPeekAtLastError() );

        // cholesky decomposition
        decompose_cholesky_kernel_device<<<batchSize, (N - row)>>>(devAs, N, row);
        gpuErrchk( hipPeekAtLastError() );

        // invert the upper
        inverse_upper_kernel_device<<<batchSize, row + 1>>>(devAs, devATmp, N, row);
        gpuErrchk( hipPeekAtLastError() );

        multiply_upper_kernel_device<<<batchSize, row + 1>>>(devATmp, devAInvs, N, row);
        gpuErrchk( hipPeekAtLastError() );
    }
    
    gpuErrchk( hipFree((void*)devATmp[0]) );
    gpuErrchk( hipHostFree((void*)devATmp) );
}

extern "C" 
void decompose_cholesky_batched_device(hipblasHandle_t handle, int N, Array *devAs, Array *devAInvs, int batchSize) {
    for (int row = 0; row < N; row++) { // loop through each row
        
        // computing diagonal elements
        pivot_cholesky_kernel_device<<<batchSize, (N - row)>>>(devAs, N, row);
        gpuErrchk( hipPeekAtLastError() );

        // cholesky decomposition
        decompose_cholesky_kernel_device<<<batchSize, (N - row)>>>(devAs, N, row);
        gpuErrchk( hipPeekAtLastError() );

    }
}



extern "C" 
void inverse_cholesky_batched_gpu(hipblasHandle_t handle, int n, Array As, Array aInvs, int batchSize) {

    Array *devAs;
    size_t pitchAs;
    Array *devAInvs;
    size_t pitchAInvs;

#ifdef DETAILED_LOGGING
    TIMER_INIT(decompose_cholesky_batched_gpu_mem_htod)
    TIMER_INIT(decompose_cholesky_batched_gpu_ker)
    TIMER_INIT(decompose_cholesky_batched_gpu_mem_dtoh)
#endif // DETAILED_LOGGING

    const size_t ArraySize = sizeof(DataType) * n * n;

    gpuErrchk( hipHostAlloc((void**)&devAs, sizeof(Array)*batchSize, hipHostMallocDefault) );
    gpuErrchk( hipHostAlloc((void**)&devAInvs, sizeof(Array)*batchSize, hipHostMallocDefault) );

    gpuErrchk( batchedCudaMalloc(devAs, &pitchAs, ArraySize, batchSize) );
    gpuErrchk( batchedCudaMalloc(devAInvs, &pitchAInvs, ArraySize, batchSize) );

#ifdef DETAILED_LOGGING
    TIMER_START(decompose_cholesky_batched_gpu_mem_htod)
#endif // DETAILED_LOGGING

    gpuErrchk( hipMemcpy2D(devAs[0], pitchAs, As, ArraySize, ArraySize, batchSize,
                hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy2D(devAInvs[0], pitchAs, As, ArraySize, ArraySize, batchSize,
                hipMemcpyHostToDevice) );

#ifdef DETAILED_LOGGING
    TIMER_STOP(decompose_cholesky_batched_gpu_mem_htod)
    TIMER_START(decompose_cholesky_batched_gpu_ker)
#endif // DETAILED_LOGGING

    inverse_cholesky_batched_device(handle, n, devAs, devAInvs, batchSize);

#ifdef DETAILED_LOGGING
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    TIMER_STOP(decompose_cholesky_batched_gpu_ker)
    TIMER_START(decompose_cholesky_batched_gpu_mem_dtoh)
#endif // DETAILED_LOGGING

    gpuErrchk( hipMemcpy2D(As, ArraySize, devAs[0], pitchAInvs, ArraySize, batchSize,
                hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy2D(aInvs, ArraySize, devAInvs[0], pitchAInvs, ArraySize, batchSize,
                hipMemcpyDeviceToHost) );

#ifdef DETAILED_LOGGING
    TIMER_STOP(decompose_cholesky_batched_gpu_mem_dtoh)

    TIMER_LOG(decompose_cholesky_batched_gpu_mem_htod)
    TIMER_LOG(decompose_cholesky_batched_gpu_ker)
    TIMER_LOG(decompose_cholesky_batched_gpu_mem_dtoh)
#endif // DETAILED_LOGGING

    gpuErrchk( hipFree((void*)devAs[0]) );
    gpuErrchk( hipFree((void*)devAInvs[0]) );
    gpuErrchk( hipHostFree((void*)devAs) );
    gpuErrchk( hipHostFree((void*)devAInvs) );
}




extern "C" void inverse_chol_gpu(Array a, int n) {
    /*
    Array a_dev;

    size_t matrixSize = n*n * sizeof(DataType);
    gpuErrchk( hipMalloc(&a_dev, matrixSize) );
    gpuErrchk( hipMemcpy(a_dev, a, matrixSize, hipMemcpyHostToDevice) );

    decomposeCholeskyGPU(a_dev, n);
    inverseLowerGPU(a_dev, n);
    multiplyLowerGPU(a_dev, n);

    gpuErrchk( hipMemcpy(a, a_dev, matrixSize, hipMemcpyDeviceToHost) );
    gpuErrchk( hipFree(a_dev) );
    */
}



int main2(int argc, char const *argv[]) {
    hipblasHandle_t handle;
    const char *directory = "tests/inverse_100_128x128";
    char filePath[1024];
    int numMatrices, m, n;
    Array a, ainv, atest;

    //Array *devAs;
    //size_t pitchAs;
    //Array *devAInvs;
    //size_t pitchAInvs;

    //struct timespec timer_start, timer_end;

    //int batchSize = 1;

    hipblasCreate(&handle);


    snprintf(filePath, 1024, "%s/a.mats", directory);
    readMatricesFile(filePath, &numMatrices, &m, &n, &a);
    readMatricesFile(filePath, &numMatrices, &m, &n, &ainv);
    //printMatrix(a, m, n);


    snprintf(filePath, 1024, "%s/ainv.mats", directory);
    readMatricesFile(filePath, &numMatrices, &m, &n, &atest);
    //printMatrix(a, m, n);

    //current_utc_time(&timer_start);
    //inverse_cholesky_batched_gpu(handle, n, a, ainv, 1);
    //printMatrix(ainv, m, n);

    inverse_cholesky_batched_gpu(handle, n, a, ainv, 1);
    //current_utc_time(&timer_end);
    //printf("New Cholesky: %fms\n", (double) (timer_end.tv_nsec - timer_start.tv_nsec) / 1000);

    //printMatrix(a, m, n);
    //printMatrix(ainv, m, n);
    /*
    // old code
    matrixSize = m * n * sizeof(DataType);

    gpuErrchk( hipMalloc(&a_dev, matrixSize) );
    gpuErrchk( hipMemcpy(a_dev, a, matrixSize, hipMemcpyHostToDevice) );

    current_utc_time(&timer_start);
    decomposeCholeskyGPU(a_dev, n);
    current_utc_time(&timer_end);
    printf("Cholesky: %fms\n", (double) (timer_end.tv_nsec - timer_start.tv_nsec) / 1000);
    //printMatrix(ainv, m, n);


    // inverse new cholesky
    //gpuErrchk( hipMemcpy(a_dev, ainv, matrixSize, hipMemcpyHostToDevice) );    

    current_utc_time(&timer_start);
    inverseLowerGPU(a_dev, n);
    current_utc_time(&timer_end);
    printf("Substitution: %fms\n", (double) (timer_end.tv_nsec - timer_start.tv_nsec) / 1000);
    
    current_utc_time(&timer_start);
    multiplyLowerGPU(a_dev, n);
    current_utc_time(&timer_end);
    printf("Multiplication: %fms\n", (double) (timer_end.tv_nsec - timer_start.tv_nsec) / 1000);

    gpuErrchk( hipMemcpy(ainv, a_dev, matrixSize, hipMemcpyDeviceToHost) );
    //printMatrix(ainv, m, n);

    //gpuErrchk( hipPeekAtLastError() );
    //gpuErrchk( hipDeviceSynchronize() );

    */

    double error = 0;
    
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            error += ainv[i * n + j] - atest[i * n + j];
        }
    }   

    printf("Error: %f\n", error);
    //printf("%fms\n", (double) (timer_end.tv_nsec - timer_start.tv_nsec) / 1000);

    return 0;
}


